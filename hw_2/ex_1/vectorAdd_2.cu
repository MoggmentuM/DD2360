
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <time.h>
#include <stdlib.h>
#include <cstdlib> //to convert char* to int
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

#define DataType double
#define max_number 100
#define minimum_number 0

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < len) out[id] = in1[id] + in2[id];
}

DataType cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp, NULL);
   return ((DataType)tp.tv_sec + (DataType)tp.tv_usec * 1.e-6);
}

//@@ Insert code to implement timer start
DataType timerStart;
DataType timerStop;

int main(int argc, char **argv) {
  srand(time(NULL));   // Initialization for random numbers, should only be called once.

  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
  // decode arguments
  if (argc < 2) {
      printf("You must provide at least one argument\n");
      exit(0);
  } else {
    inputLength = atoi(argv[1]);
  }

  printf("The input length is %d\n", inputLength);

  size_t size = inputLength * sizeof(DataType);

  //@@ Insert code below to allocate Host memory for input and output
  hostInput1 = (DataType *)malloc(size);
  hostInput2 = (DataType *)malloc(size);
  hostOutput = (DataType *)malloc(size);

  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  for (int i = 0; i < inputLength; i++) {
    // generate a pseudo-random integer between minimum_number and max_number
    hostInput1[i] = rand() % (max_number + 1 - minimum_number) + minimum_number;
    hostInput2[i] = rand() % (max_number + 1 - minimum_number) + minimum_number;
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc((void **)&deviceInput1, size);
  hipMalloc((void **)&deviceInput2, size);
  hipMalloc((void **)&deviceOutput, size);

  //@@ Insert code to below to Copy memory to the GPU here
  // Timer Start
  timerStart = cpuSecond();

  hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);

  // Timer Stop
  timerStop = cpuSecond();
  DataType copyToDeviceTime = timerStop - timerStart;
  printf("Data copy from Host to Device elapsed %f sec\n", copyToDeviceTime);

  //@@ Initialize the 1D grid and block dimensions here
  int dimx = 32;
  dim3 block(dimx, 1);
  dim3 grid((inputLength + block.x - 1) / block.x, 1);

  //@@ Launch the GPU Kernel here
  // Timer Start
  timerStart = cpuSecond();

  vecAdd<<<grid, block>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
  hipDeviceSynchronize();

  // Timer Stop
  timerStop = cpuSecond();
  DataType kernelTime = timerStop - timerStart;
  printf("CUDA Kernel elapsed %f sec\n", kernelTime);

  //@@ Copy the GPU memory back to the CPU here
  // Timer Start
  timerStart = cpuSecond();

  hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);

  // Timer Stop
  timerStop = cpuSecond();
  DataType copyToHostTime = timerStop - timerStart;
  printf("Data copy from Device to Host elapsed %f sec\n", copyToHostTime);

  //@@ Insert code below to compare the output with the reference
  double tolerance = 1.0e-14;
  DataType expected;
  for (int i = 0; i < inputLength; i++) {
    expected = hostInput1[i] + hostInput2[i];
    // if the absolute value is greater than the tolerance we have an error
    if (fabs(hostOutput[i] - expected) > tolerance) {
      printf("\nError: value of hostOutput[%d] = %f instead of %f\n\n", i, hostOutput[i], expected);
      exit(1);
    } else {
      // printf("\nOk: value of hostOutput[%d] = %f - expected: %f\n\n", i, hostOutput[i], expected);
    }
  }

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  printf("\n---------------------------------------------\n");
  printf("SUCCESS\n");

  return 0;
}
